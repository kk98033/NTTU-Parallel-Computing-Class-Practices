#include <hip/hip_runtime.h>
#include <stdio.h>
void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }
    return;
}
void initialData(float *ip, int size)
{
    int i;
    for (i = 0; i < size; i++)
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    return;
}

void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
    for (int idx = 0; idx < N; idx++)
    {
        C[idx] = A[idx] + B[idx];
    }
}
__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}
__global__ void sumArraysZeroCopy(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

int main(int argc, char **argv)
{
    int dev = 0;
    hipSetDevice(dev);
    hipEvent_t time1, time2, time3, time4;
    float kernelExecutionTime, MemcpyfromHtoDTime, MemcpyfromDtoHTime, TotalTime;
    double start, elapsed_time;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    // check if support mapped memory
    if (!deviceProp.canMapHostMemory)
    {
        printf("Device %d does not support mapping CPU host memory!\n", dev);
        hipDeviceReset();
        exit(EXIT_SUCCESS);
    }
    printf("Using Device %d: %s ", dev, deviceProp.name);
    hipEventCreate(&time1);
    hipEventCreate(&time2);
    hipEventCreate(&time3);
    hipEventCreate(&time4);

    int ipower = 10; // set up data size of vectors
    if (argc > 1)
        ipower = atoi(argv[1]);
    int nElem = 1 << ipower;
    size_t nBytes = nElem * sizeof(float);
    if (ipower < 18)
    {
        printf("Vector size %d power %d nbytes %3.0f KB\n", nElem, ipower,
               (float)nBytes / (1024.0f));
    }
    else
    {
        printf("Vector size %d power %d nbytes %3.0f MB\n", nElem, ipower,
               (float)nBytes / (1024.0f * 1024.0f));
    }
    // malloc host memory
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);
    initialData(h_A, nElem);
    initialData(h_B, nElem);
    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);
    hipEventRecord(time1, 0);
    sumArraysOnHost(h_A, h_B, hostRef, nElem);
    hipEventRecord(time2, 0);
    hipEventSynchronize(time1);
    hipEventSynchronize(time2);
    hipEventElapsedTime(&kernelExecutionTime,
                         time1, time2);
    printf("cpu computation: %20.5f ms\n",
           kernelExecutionTime);
    // 配置在device上的一般global memory變數
    float *d_A, *d_B, *d_C;
    hipMalloc((float **)&d_A, nBytes);
    hipMalloc((float **)&d_B, nBytes);
    hipMalloc((float **)&d_C, nBytes);

    int iLen = 512;
    dim3 block(iLen);
    dim3 grid((nElem + block.x - 1) / block.x);
    hipEventRecord(time1, 0);
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
    hipEventRecord(time2, 0);
    sumArrays<<<grid, block>>>(d_A, d_B, d_C, nElem);
    hipEventRecord(time3, 0);
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);
    hipEventRecord(time4, 0);
    hipEventSynchronize(time1);
    hipEventSynchronize(time2);
    hipEventSynchronize(time3);
    hipEventSynchronize(time4);
    hipEventElapsedTime(&MemcpyfromHtoDTime, time1, time2);
    hipEventElapsedTime(&kernelExecutionTime, time2, time3);
    hipEventElapsedTime(&MemcpyfromDtoHTime, time3, time4);
    hipEventElapsedTime(&TotalTime, time1, time4);
    printf("Normal(Pageabled memory) H->D: %7.4f ms, kernel computation: %7.4f ms, D->H: %7.4f ms, TOTAL: %7.4f ms\n", MemcpyfromHtoDTime, kernelExecutionTime,
           MemcpyfromDtoHTime, TotalTime);

    // check device results
    checkResult(hostRef, gpuRef, nElem);
    // free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    // free host memory
    free(h_A);
    free(h_B);
    free(gpuRef);
    // part 2: 使用zero-copy memory
    // 配置zero-copy memory
    hipHostAlloc((void **)&h_A, nBytes, hipHostMallocMapped);
    hipHostAlloc((void **)&h_B, nBytes, hipHostMallocMapped);
    hipHostAlloc((void **)&gpuRef, nBytes, hipHostMallocMapped);
    initialData(h_A, nElem);
    initialData(h_B, nElem);
    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);
    // add at host side for result checks
    sumArraysOnHost(h_A, h_B, hostRef, nElem);

    // device pointer與host pointer的zero-copy對應
    hipHostGetDevicePointer((void **)&d_A, (void *)h_A, 0);
    hipHostGetDevicePointer((void **)&d_B, (void *)h_B, 0);
    hipHostGetDevicePointer((void **)&d_C, (void *)gpuRef, 0);
    hipEventRecord(time1, 0);
    // 直接使用上述的device指標執行kernel函式，不用透過cudaMemcpy作h->d的資料傳送
    sumArraysZeroCopy<<<grid, block>>>(d_A, d_B, d_C, nElem);
    hipEventRecord(time2, 0);
    hipEventSynchronize(time1);
    hipEventSynchronize(time2);
    hipEventElapsedTime(&kernelExecutionTime, time1, time2);
    printf("Zerocopy(Pinned memory) kernel computation: %7.4f ms\n", kernelExecutionTime);
    // check device results
    checkResult(hostRef, gpuRef, nElem);
    // CHECK(cudaFreeHost(h_A));
    // CHECK(cudaFreeHost(h_B));
    // CHECK(cudaFreeHost(gpuRef));
    free(hostRef);
    hipDeviceReset();
    return EXIT_SUCCESS;
}