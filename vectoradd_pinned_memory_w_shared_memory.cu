#include <stdio.h>
#include <hip/hip_runtime.h>
// Kernel definition
__global__ void VecAdd(float *A, float *B, float *C)
{
    int index = threadIdx.x;

    // 修改處
    extern __shared__ float shared[];
    float *s_A = shared;
    float *s_B = &shared[blockDim.x];
    s_A[index] = A[index];
    s_B[index] = B[index];
    __syncthreads();
    C[index] = s_A[index] + s_B[index];
}

int main()
{
    int N = 1 << 10;
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;
    hipEvent_t time1, time2;
    float kernelExecutionTime;

    // 0d grid of 1D block
    dim3 dimblock(1024, 1, 1);
    dim3 dimgrid(1, 1, 1);

    printf("dimblock.x = %d, dimblock.y = %d, dimblock.z = %d\n", dimblock.x,
           dimblock.y, dimblock.z);
    printf("dimgrid.x = %d, dimgrid.y = %d, dimgrid.z = %d\n", dimgrid.x,
           dimgrid.y, dimgrid.z);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using Device %d: %s\n", dev,
           deviceProp.name);

    // 修改處 
    // Allocate pinned memory for host vectors
    hipHostAlloc((void **)&h_A, N * sizeof(float), hipHostMallocDefault);
    hipHostAlloc((void **)&h_B, N * sizeof(float), hipHostMallocDefault);
    hipHostAlloc((void **)&h_C, N * sizeof(float), hipHostMallocDefault);

    // 修改處
    // Initialize host memory
    for (int i = 0; i < N; i++)
    {
        h_A[i] = 20.0f;
        h_B[i] = 30.0f;
    }
    
    // 修改處
    // Allocate memory for device vectors
    hipMalloc((void **)&d_A, N * sizeof(float));
    hipMalloc((void **)&d_B, N * sizeof(float));
    hipMalloc((void **)&d_C, N * sizeof(float));

    // 修改處
    // Copy data from host to device
    hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice);

    hipEventCreate(&time1);
    hipEventCreate(&time2);

    hipEventRecord(time1, 0);

    // 修改處
    VecAdd<<<dimgrid, dimblock, 2 * dimblock.x * sizeof(float)>>>(d_A, d_B, d_C);
    hipEventRecord(time2, 0);

    hipEventSynchronize(time1);
    hipEventSynchronize(time2);

    hipEventElapsedTime(&kernelExecutionTime, time1, time2);

    hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    printf("h_C[%d] == %7.2f\n", N - 1, h_C[N - 1]);
    printf("Kernel Execution = %7.2f ms\n", kernelExecutionTime);

    // 修改處
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    return (0);
}