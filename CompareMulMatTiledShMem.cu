/*
./CompareMulMatTiledShMem 
N = 64, TILE_WIDTH = 16
dimGrid.x = 4, dimGrid.y = 4
MulMatCPU			 elapsed time: 0.696287989616394043ms
MulMatTiledShMem		 elapsed time: 0.017408000305294991 ms
MulMatTiledShMemNoBankConflicts	 elapsed time: 0.007615999784320593 ms
simpleMatMulKernel		 elapsed time: 0.005024000070989132 ms
Results Matched!
h_C[0][0] = 87360
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#define TILE_WIDTH 16
void MulMatCPU(uint *A, uint *B, uint *C, uint N)
{
    for (int i = 0; i < N; i++)
        for (int k = 0; k < N; k++)
            for (int j = 0; j < N; j++)
                C[i * N + j] += A[i * N + k] * B[k * N + j];
}

/* 
## Practice

 實作一簡易矩陣相乘的kernel函式，支援一維的
grid（即blockDim.x>=1), 並和本節其它矩陣相
乘方法所需時間作比較。
*/
__global__ void simpleMatMulKernel(uint *A, uint *Nd, uint *Pd, int Width) {
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;

    if (tx >= Width || ty >= Width) return;

    uint Pvalue = 0;
    for (int k = 0; k < Width; k++) {
        Pvalue += A[ty * Width + k] * Nd[k * Width + tx];
    }
    Pd[ty * Width + tx] = Pvalue;
}

__global__ void MulMatTiledShMem(uint *Md, uint *Nd, uint *Pd, int Width)
{
    __shared__ uint Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ uint Nds[TILE_WIDTH][TILE_WIDTH];
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;
    uint Pvalue = 0;
    // 反覆計算Pd[Row][Col]所需之Md 和Nd tiles的資料，並計算乘積的迴圈
    for (int m = 0; m < Width / TILE_WIDTH; m++)
    {
        Mds[ty][tx] = Md[Row * Width + (m * TILE_WIDTH + tx)];
        Nds[ty][tx] = Nd[Col + (m * TILE_WIDTH + ty) * Width];
        __syncthreads();
        for (int k = 0; k < TILE_WIDTH; k++) // 小塊矩陣相乘運算
            Pvalue += Mds[ty][k] * Nds[k][tx];
        __syncthreads();
    }
    Pd[Row * Width + Col] = Pvalue; // 將Pd[Row][Col]所需之各小塊結果累加得到最終結果
}

__global__ void MulMatTiledShMemNoBankConflicts(uint *Md, uint *Nd, uint *Pd, int Width)
{
    __shared__ uint Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ uint Nds[TILE_WIDTH][TILE_WIDTH];
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;
    uint Pvalue = 0;
    for (int m = 0; m < Width / TILE_WIDTH; m++)
    {
        Mds[ty][tx] = Md[Col * Width + (m * TILE_WIDTH + ty)];
        Nds[ty][tx] = Nd[Row + (m * TILE_WIDTH + tx) * Width];
        __syncthreads();
        for (int k = 0; k < TILE_WIDTH; k++)
            Pvalue += Mds[k][tx] * Nds[ty][k];
        __syncthreads();
    }
    Pd[Col * Width + Row] = Pvalue;
}

int main(int argc, char **argv)
{
    int N = 64;
    if (argc == 2)
        N = 1 << atoi(argv[1]);
    printf("N = %d, TILE_WIDTH = %d\n",
           N, TILE_WIDTH);
    hipEvent_t time1, time2;
    float kernelExecutionTime;
    uint *h_A, *h_B, *h_C, *cpu_C;
    uint *d_A, *d_B, *d_C;

    h_A = (uint *)malloc(N * N * sizeof(uint));
    h_B = (uint *)malloc(N * N * sizeof(uint));
    h_C = (uint *)malloc(N * N * sizeof(uint));
    cpu_C = (uint *)malloc(N * N * sizeof(uint));

    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
        {
            h_A[i * N + j] = i + j;
            h_B[i * N + j] = i + j + 1;
            h_C[i * N + j] = 0;
            cpu_C[i * N + j] = 0;
        }

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid(N / TILE_WIDTH, N / TILE_WIDTH);

    printf("dimGrid.x = %d, dimGrid.y = %d\n",
           dimGrid.x, dimGrid.y);

    hipEventCreate(&time1);
    hipEventCreate(&time2);

    hipEventRecord(time1, 0);
    MulMatCPU(h_A, h_B, cpu_C, N);
    hipEventRecord(time2, 0);

    hipEventSynchronize(time1);
    hipEventSynchronize(time2);
    hipEventElapsedTime(&kernelExecutionTime,
                         time1, time2);
    printf("MulMatCPU\t\t\t elapsed time: %20.18fms\n", kernelExecutionTime);
    
    hipMalloc((void **)&d_A, N * N * sizeof(uint));
    hipMalloc((void **)&d_B, N * N * sizeof(uint));
    hipMalloc((void **)&d_C, N * N * sizeof(uint));

    hipMemcpy(d_A, h_A, N * N * sizeof(uint),
               hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(uint),
               hipMemcpyHostToDevice);
    hipEventRecord(time1, 0);
    
    MulMatTiledShMem<<<dimGrid,
                       dimBlock>>>(d_A, d_B, d_C, N);
    hipEventRecord(time2, 0);

    hipEventSynchronize(time1);
    hipEventSynchronize(time2);

    hipEventElapsedTime(&kernelExecutionTime, time1, time2);
    printf("MulMatTiledShMem\t\t elapsed time: %20.18f ms\n", kernelExecutionTime);
    hipMemcpy(h_C, d_C, N * N * sizeof(uint),
               hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
            if (h_C[i * N + j] != cpu_C[i * N + j])
            {
                printf("h_C[%d][%d]==%d != cpu_C[%d][%d]==%d\n", i, j, i, j, h_C[i * N + j], cpu_C[i * N + j]);
                return -1;
            }
    hipEventRecord(time1, 0);
    MulMatTiledShMemNoBankConflicts<<<dimGrid,
                                      dimBlock>>>(d_A, d_B, d_C, N);
    hipEventRecord(time2, 0);
    hipEventSynchronize(time1);
    hipEventSynchronize(time2);
    hipEventElapsedTime(&kernelExecutionTime,
                         time1, time2);
    printf("MulMatTiledShMemNoBankConflicts\t elapsed time: %20.18f ms\n", kernelExecutionTime);
    hipMemcpy(h_C, d_C, N * N * sizeof(uint),
               hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
            if (h_C[i * N + j] != cpu_C[i * N + j])
            {
                printf("h_C[%d][%d]==%d != cpu_C[%d][%d]==%d\n", i, j, i, j, h_C[i * N + j], cpu_C[i * N + j]);
                return -1;
            }

    /*
        ## Practice

         實作一簡易矩陣相乘的kernel函式，支援一維的
        grid（即blockDim.x>=1), 並和本節其它矩陣相
        乘方法所需時間作比較。
    */
    dim3 dimBlockSimple(256);
    dim3 dimGridSimple((N + 255) / 256);

    hipEventRecord(time1, 0);
    simpleMatMulKernel<<<dimGridSimple, dimBlockSimple>>>(d_A, d_B, d_C, N);
    hipEventRecord(time2, 0);

    hipEventSynchronize(time1);
    hipEventSynchronize(time2);
    hipEventElapsedTime(&kernelExecutionTime, time1, time2);
    printf("simpleMatMulKernel\t\t elapsed time: %20.18f ms\n", kernelExecutionTime);
    
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
            if (h_C[i * N + j] != cpu_C[i * N + j])
            {
                printf("h_C[%d][%d]==%d != cpu_C[%d][%d]==%d\n", i, j, i, j, h_C[i * N + j], cpu_C[i * N + j]);
                return -1;
            }


    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    printf("Results Matched!\n");
    printf("h_C[0][0] = %d\n", h_C[0]);
    free(h_A);
    free(h_B);
    free(h_C);
    free(cpu_C);
    return 0;
}