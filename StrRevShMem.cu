/*
./StrRevShMem 
RevShmem elapsed time: 1.536000013351440430 ms
a[0]=  0, a[1]=  1, a[2]=  2, 
d[0]= 63, d[1]= 62, d[2]= 61, 

*/

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void RevShmem(int *d, int n)
{
    extern __shared__ int s[]; // 動態共享記憶體
    int t = threadIdx.x; // 定義每個執行緒的索引
    int tr = n - t - 1;  // 計算反轉的索引
    s[t] = d[t]; // 將全域記憶體的資料複製到共享記憶體
    __syncthreads(); // 確保所有數據都已寫入共享記憶體
    d[t] = s[tr]; // 從共享記憶體複製反轉後的資料回全域記憶體
}

int main(void)
{
    const int n = 64;
    int a[n], r[n], d[n];
    hipEvent_t time1, time2;
    float kernelExecutionTime;
    hipEventCreate(&time1);
    hipEventCreate(&time2);
    for (int i = 0; i < n; i++)
    {
        a[i] = i;
        r[i] = n - i - 1;
        d[i] = 0;
    }
    int *d_d;
    hipMalloc(&d_d, n * sizeof(int));
    hipMemcpy(d_d, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(time1, 0);
    // 動態共享記憶體
    RevShmem<<<1, n, n * sizeof(int)>>>(d_d, n);
    hipEventRecord(time2, 0);
    hipMemcpy(d, d_d, n * sizeof(int), hipMemcpyDeviceToHost);

    hipEventSynchronize(time1);
    hipEventSynchronize(time2);
    hipEventElapsedTime(&kernelExecutionTime, time1, time2);
    printf("RevShmem elapsed time: %20.18f ms\n", kernelExecutionTime);
    for (int i = 0; i < n; i++)
        if (d[i] != r[i])
            printf("Error: d[%d]!=r[%d] (%d, %d)\n", i, i, d[i], r[i]);
    for (int i = 0; i < 3; i++)
        printf("a[%d]=%3d, ", i, a[i]);
    printf("\n");
    for (int i = 0; i < 3; i++)
        printf("d[%d]=%3d, ", i, d[i]);
    printf("\n");

    hipFree(d_d);
    hipDeviceReset();
    return EXIT_SUCCESS;
}
