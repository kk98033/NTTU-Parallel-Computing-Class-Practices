/*
./pageable_memTransfer 
./pageable_memTransfer starting at device 0: NVIDIA GeForce RTX 3050 Ti Laptop GPU memory size 4194304 nbyte 16.00MB

*/

#include <hip/hip_runtime.h>
#include <stdio.h>
int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipSetDevice(dev);
    // memory size
    unsigned int isize = 1 << 22;
    unsigned int nbytes = isize * sizeof(float);
    // get device information
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("%s starting at ", argv[0]);
    printf("device %d: %s memory size %d nbyte %5.2fMB\n", dev,
           deviceProp.name, isize, nbytes / (1024.0f * 1024.0f));
    // allocate the host memory
    float *h_a = (float *)malloc(nbytes);
    // allocate the device memory
    float *d_a;
    hipMalloc((float **)&d_a, nbytes);
    // initialize the host memory
    for (unsigned int i = 0; i < isize; i++)
        h_a[i] = 0.5f;
    // transfer data from the host to the device
    hipMemcpy(d_a, h_a, nbytes,
               hipMemcpyHostToDevice);
    // transfer data from the device to the host
    hipMemcpy(h_a, d_a, nbytes,
               hipMemcpyDeviceToHost);
    hipFree(d_a);
    free(h_a);
    hipDeviceReset();
    return EXIT_SUCCESS;
}

