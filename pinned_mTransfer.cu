/*
./pinned_mTransfer 
./pinned_mTransfer starting at device 0: NVIDIA GeForce RTX 3050 Ti Laptop GPU memory size 4194304 nbyte 16.00MB

*/

#include <hip/hip_runtime.h>
#include <stdio.h>
int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipSetDevice(dev);
    // memory size
    unsigned int isize = 1 << 22;
    unsigned int nbytes = isize * sizeof(float);
    // get device information
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("%s starting at ", argv[0]);
    printf("device %d: %s memory size %d nbyte %5.2fMB\n", dev,
           deviceProp.name, isize, nbytes / (1024.0f * 1024.0f));
    // allocate the pinned memory
    float *h_aPinned;
    hipError_t status =
        hipHostMalloc((void **)&h_aPinned, nbytes, hipHostMallocDefault);
    if (status != hipSuccess)
    {
        fprintf(stderr, "Error returned from pinned host memory allocation\n");
        exit(1);
    } // allocate the device memory
    float *d_a;
    hipMalloc((float **)&d_a, nbytes);
    // initialize the host memory
    for (unsigned int i = 0; i < isize; i++)
        h_aPinned[i] = 0.5f;
    // transfer data from the host to the device
    hipMemcpy(d_a, h_aPinned, nbytes,
               hipMemcpyHostToDevice);
    // transfer data from the device to the host
    hipMemcpy(h_aPinned, d_a, nbytes,
               hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipHostFree(h_aPinned);
    hipDeviceReset();
    return EXIT_SUCCESS;
}